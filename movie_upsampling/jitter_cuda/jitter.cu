#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define H_THREADS 32
#define W_THREADS 32


template<typename scalar_t>
__global__ void _kern_jitter_frames_forward(
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> frames,
        const torch::PackedTensorAccessor<int64_t, 3, torch::RestrictPtrTraits, size_t> jitter_coords,
        torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> jitter_dest) {

    const int64_t height = frames.size(1);
    int64_t h_index = threadIdx.x;
    int64_t h_stride = blockDim.x;

    const int64_t width = frames.size(2);
    int64_t w_index = threadIdx.y;
    int64_t w_stride = blockDim.y;

    int64_t b = blockIdx.x;
    int64_t f = blockIdx.y;

    /*
    const int64_t batch = jitter_dest.size(0);
    const int64_t n_frames = jitter_dest.size(1);
    int64_t b = blockIdx.x / n_frames;
    int64_t f = blockIdx.x % n_frames;

    const int64_t height = frames.size(1);
    int64_t h_index = threadIdx.y + blockIdx.y * blockDim.y;
    int64_t h_stride = blockDim.y * gridDim.y;

    const int64_t width = frames.size(2);
    int64_t w_index = threadIdx.z + blockIdx.z * blockDim.z;
    int64_t w_stride = blockDim.z * gridDim.z;
     */

    int64_t jitter_h = jitter_coords[b][f][0];
    int64_t jitter_w = jitter_coords[b][f][1];

    scalar_t ZERO = 0.0;

    for (int64_t h = h_index; h < height; h += h_stride) {
        int64_t source_h = h - jitter_h;
        bool valid_source_h = (source_h >= 0) && (source_h < height);

        for (int64_t w = w_index; w < width; w += w_stride) {
            int64_t source_w = w - jitter_w;
            bool valid_source_w = (source_w >= 0) && (source_w < width);

            jitter_dest[b][f][h][w] = (valid_source_h && valid_source_w) ? frames[b][source_h][source_w] : ZERO;
        }
    }
}


torch::Tensor _jitter_frames_forward(torch::Tensor frames,
                                    torch::Tensor jitter_coords) {
    /*
     *parallelize by height and width, since there will be many pixels, but probably not
     * very many frames
     */

    const int64_t batch = frames.size(0);
    const int64_t height = frames.size(1);
    const int64_t width = frames.size(2);
    const int64_t n_jitter_frames = jitter_coords.size(1);

    auto options = torch::TensorOptions()
            .dtype(frames.dtype())
            .layout(torch::kStrided)
            .device(frames.device());
    torch::Tensor dest = torch::empty(std::vector<int64_t>({batch, n_jitter_frames, height, width}), options);

    /*
    const dim3 threads(1, 16, 16);
    const dim3 blocks(batch * n_jitter_frames, 4, 4);
     */
    const dim3 threads(16, 16);
    const dim3 blocks(batch, n_jitter_frames);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(dest.scalar_type(), "_kern_jitter_frames_forward", [&] {
        _kern_jitter_frames_forward<scalar_t><<<blocks, threads>>>(
                frames.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                jitter_coords.packed_accessor<int64_t, 3, torch::RestrictPtrTraits, size_t>(),
                dest.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>());
    });

    return dest;
}


template<typename scalar_t>
__global__ void _kern_jitter_frames_backward(
        const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> d_output_d_jittered_frames,
        const torch::PackedTensorAccessor<int64_t, 3, torch::RestrictPtrTraits, size_t> jitter_coords,
        torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> d_output_d_static_frames) {

    int64_t batch = d_output_d_static_frames.size(0);

    int64_t n_frames = jitter_coords.size(1);

    int64_t b = blockIdx.x;

    int64_t height = d_output_d_static_frames.size(1);
    int64_t h_index = threadIdx.x;
    int64_t h_stride = blockDim.x;

    int64_t width = d_output_d_static_frames.size(2);
    int64_t w_index = threadIdx.y;
    int64_t w_stride = blockDim.y;

    scalar_t ZERO = 0.0;

    for (int64_t f = 0; f < n_frames; ++f) {
        int64_t offset_h = jitter_coords[b][f][0];
        int64_t offset_w = jitter_coords[b][f][1];
        for (int64_t h = h_index; h < height; h += h_stride) {
            int64_t read_h = h + offset_h;
            bool sat_h = (read_h >= 0) && (read_h < height);
            for (int64_t w = w_index; w < width; w += w_stride) {

                int64_t read_w = w + offset_w;

                bool sat_w = (read_w >= 0) && (read_w < width);

                scalar_t add_to = (sat_h && sat_w) ? d_output_d_jittered_frames[b][f][read_h][read_w] : ZERO;
                d_output_d_static_frames[b][h][w] += add_to;
            }
        }
    }
}


torch::Tensor _jitter_frames_backward(torch::Tensor d_output_d_jittered_frames,
                                     torch::Tensor jitter_coords) {
    /*
     * Computes the gradient of the loss with respect to the input
     */

    int64_t batch = d_output_d_jittered_frames.size(0);
    int64_t n_jittered_frames = d_output_d_jittered_frames.size(1);
    int64_t height = d_output_d_jittered_frames.size(2);
    int64_t width = d_output_d_jittered_frames.size(3);

    auto options = torch::TensorOptions()
            .dtype(d_output_d_jittered_frames.dtype())
            .layout(torch::kStrided)
            .device(d_output_d_jittered_frames.device());
    torch::Tensor dest = torch::zeros(std::vector<int64_t>({batch, height, width}), options);

    const dim3 threads(32, 32);
    const dim3 blocks(batch);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(dest.scalar_type(), "_kern_jitter_frames_backward", [&] {
        _kern_jitter_frames_backward<scalar_t><<<blocks, threads>>>(
                d_output_d_jittered_frames.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
                jitter_coords.packed_accessor<int64_t, 3, torch::RestrictPtrTraits, size_t>(),
                dest.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>());
    });

    return dest;
}
