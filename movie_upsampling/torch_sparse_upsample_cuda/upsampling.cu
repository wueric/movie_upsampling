#include "hip/hip_runtime.h"
#define INVALID_IDX -1

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>

template<typename scalar_t>
__global__ void dumb_add_kernel(
        const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> a,
        const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> b,
        torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dest) {

    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t stride = blockDim.x * gridDim.x;

    const int64_t max_N = a.size(0);
    for (int64_t i = index; i < max_N; i += stride) {
        dest[i] = a[i] + b[i];
    }
}

torch::Tensor dumb_add_cuda(torch::Tensor a_tens,
                            torch::Tensor b_tens) {

    int64_t dim_a = a_tens.size(0);

    auto options = torch::TensorOptions()
            .dtype(a_tens.dtype())
            .layout(torch::kStrided)
            .device(a_tens.device());

    torch::Tensor dest = torch::empty(std::vector<int64_t>({dim_a}), options);

    const int threads = 1024;
    const dim3 blocks((dim_a + threads - 1) / threads);

    AT_DISPATCH_FLOATING_TYPES(dest.scalar_type(), "dumb_add", [&] {
        dumb_add_kernel<scalar_t><<<blocks, threads>>>(
            a_tens.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            b_tens.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            dest.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>());
    });

    return dest;
}

template<typename scalar_t>
__global__ void sparse_time_domain_movie_upsample_kernel(
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> movie_frames,
        torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> us_dest,
        const torch::PackedTensorAccessor<int64_t, 2, torch::RestrictPtrTraits, size_t> frame_selection,
        const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> frame_weights) {

    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t stride = blockDim.x * gridDim.x;

    const int64_t max_N = us_dest.size(0);
    const int64_t height = us_dest.size(1);
    const int64_t width = us_dest.size(2);

    for (int64_t i = index; i < max_N; i += stride) {
        if (frame_selection[i][1] == INVALID_IDX) {
            int64_t only_frame_ix = frame_selection[i][0];
            for (int64_t h = 0; h < height; ++h) {
                for (int64_t w = 0; w < width; ++w) {
                    us_dest[i][h][w] = movie_frames[only_frame_ix][h][w];
                }
            }
        } else {

            int64_t first_frame_ix = frame_selection[i][0];
            scalar_t first_weight = frame_weights[i][0];

            int64_t second_frame_ix = frame_selection[i][1];;
            scalar_t second_weight = frame_weights[i][1];
            for (int64_t h = 0; h < height; ++h) {
                for (int64_t w = 0; w < height; ++w) {
                    scalar_t first_val = movie_frames[first_frame_ix][h][w];
                    scalar_t second_val = movie_frames[second_frame_ix][h][w];

                    scalar_t write_val = first_val * first_weight + second_val * second_weight;
                    us_dest[i][h][w] = write_val;
                }
            }
        }
    }
}

torch::Tensor upsample_sparse_movie_cuda(torch::Tensor movie_frames,
                                         torch::Tensor frame_selection,
                                         torch::Tensor frame_weights) {
    int64_t n_bins = frame_selection.size(0);
    int64_t height = movie_frames.size(1);
    int64_t width = movie_frames.size(2);

    auto options = torch::TensorOptions()
            .dtype(frame_weights.dtype())
            .layout(torch::kStrided)
            .device(movie_frames.device());

    torch::Tensor dest = torch::empty(std::vector<int64_t>({n_bins, height, width}), options);

    const int threads = 1024;
    const dim3 blocks((n_bins + threads - 1) / threads);

    AT_DISPATCH_FLOATING_TYPES(dest.scalar_type(), "sparse_upsample_movie", [&] {
        sparse_time_domain_movie_upsample_kernel<scalar_t><<<blocks, threads>>>(
                movie_frames.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                dest.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                frame_selection.packed_accessor<int64_t, 2, torch::RestrictPtrTraits, size_t>(),
                frame_weights.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>());
    });

    return dest;
}
